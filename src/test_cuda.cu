#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define ROW_NUM 10000100
#define COLUMN_NUM 4

__global__
void request(int *tab, int *result)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < ROW_NUM; i += stride)
    {
        if(result[i] > 1000 && result[i] < 100000
        && result[i+ROW_NUM] > 1000 && result[i+ROW_NUM] < 100000
        && result[i+ROW_NUM*2] > 1000 && result[i+ROW_NUM*2] < 100000)
        {
            result[i] = 1;
        }
        else
        {
            result[i] = 0;
        }
    }
}

int main(void)
{
    int *tab, *result;
    
    hipMallocManaged(&tab, ROW_NUM*COLUMN_NUM*sizeof(int));
    hipMallocManaged(&result, ROW_NUM*sizeof(int));
    
    srand(0);
    
    for(int column=0;column<COLUMN_NUM-1;++column)
    {
        for(int row=0;row<ROW_NUM;++row)
        {
            tab[ROW_NUM*column+row] = rand()%1000000;
        }
    }
    
    //t1 = myCPUTimer();
    
    request<<<(ROW_NUM+255)/256, 256>>>(tab, result);
    
    hipDeviceSynchronize();
    
    //t2 = myCPUTimer();
    
    int total = 0;
    for(int row=0;row<ROW_NUM;++row)
    {
        if(result[i])
        {
            ++total;
        }
    }
    std::cout << "Total : " << total << std::endl;
    
    hipFree(tab);
    hipFree(result);
      
    return 0;
}