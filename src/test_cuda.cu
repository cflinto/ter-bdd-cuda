
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define ROW_NUM 10000100
#define COLUMN_NUM 4

__global__
void request(int *tab, int *result)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < ROW_NUM; i += stride)
    {
        if(tab[i] > 1000 && tab[i] < 100000
        && tab[i+ROW_NUM] > 1000 && tab[i+ROW_NUM] < 100000
        && tab[i+ROW_NUM*2] > 1000 && tab[i+ROW_NUM*2] < 100000)
        {
            result[i] = 1;
        }
        else
        {
            result[i] = 0;
        }
    }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

int main(void)
{
    int *tab, *result; // GPU
    int *tabCPU, *resultCPU; // CPU
    
    tabCPU = new int[ROW_NUM*COLUMN_NUM];
    resultCPU = new int[ROW_NUM];
    
    hipMalloc(&tab, ROW_NUM*COLUMN_NUM*sizeof(int));
    hipMalloc(&result, ROW_NUM*sizeof(int));
    
    srand(0);
    
    for(int column=0;column<COLUMN_NUM-1;++column)
    {
        for(int row=0;row<ROW_NUM;++row)
        {
            tabCPU[ROW_NUM*column+row] = rand()%1000000;
        }
    }
    
    hipMemcpy(tab, tabCPU, ROW_NUM*COLUMN_NUM*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(result, resultCPU, ROW_NUM*sizeof(int), hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    request<<<(ROW_NUM+255)/256, 256>>>(tab, result);
    
    hipEventRecord(stop);
    
    hipMemcpy(resultCPU, result, ROW_NUM*sizeof(int), hipMemcpyDeviceToHost);
    
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    /*
    gpuErrchk( cudaPeekAtLastError() );
    gpuErrchk( cudaDeviceSynchronize());
    */
    
    
    int total = 0;
    for(int row=0;row<ROW_NUM;++row)
    {
        if(result[row])
        {
            ++total;
        }
    }
    std::cout << "Total : " << total << std::endl;
    
    
    std::cout << milliseconds;
    
    hipFree(tab);
    hipFree(result);
    
    delete[] resultCPU;
    delete[] tabCPU;
      
    return 0;
}