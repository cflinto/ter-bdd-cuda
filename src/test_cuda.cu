#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define ROW_NUM 10000100
#define COLUMN_NUM 4

__global__
void request(int *tab, int *result)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < ROW_NUM; i += stride)
    {
        if(tab[i] > 1000 && tab[i] < 100000
        && tab[i+ROW_NUM] > 1000 && tab[i+ROW_NUM] < 100000
        && tab[i+ROW_NUM*2] > 1000 && tab[i+ROW_NUM*2] < 100000)
        {
            result[i] = 1;
        }
        else
        {
            result[i] = 0;
        }
    }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

int main(void)
{
    /*int *tab, *result; // GPU
    int *tabCPU, *resultCPU; // CPU
    
    tabCPU = (int*)malloc(ROW_NUM*COLUMN_NUM*sizeof(int));
    resultCPU = (int*)malloc(ROW_NUM*sizeof(int));
    
    hipMalloc(&tab, ROW_NUM*COLUMN_NUM*sizeof(int));
    hipMalloc(&result, ROW_NUM*sizeof(int));
    
    srand(0);
    
    for(int column=0;column<COLUMN_NUM-1;++column)
    {
        for(int row=0;row<ROW_NUM;++row)
        {
            tabCPU[ROW_NUM*column+row] = rand()%1000000;
        }
    }
    
    hipMemcpy(tab, tabCPU, ROW_NUM*COLUMN_NUM*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(result, resultCPU, ROW_NUM*sizeof(int), hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    request<<<(ROW_NUM+255)/256, 256>>>(tab, result);
    
    hipEventRecord(stop);
    
    hipMemcpy(resultCPU, result, ROW_NUM*sizeof(int), hipMemcpyDeviceToHost);
    
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);*/
    
    /*
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize());
    */
    
    
    /*int total = 0;
    for(int row=0;row<ROW_NUM;++row)
    {
        if(result[row])
        {
            ++total;
        }
    }
    std::cout << "Total : " << total << std::endl;
    
    
    std::cout << milliseconds;
    
    hipFree(tab);
    hipFree(result);
    
    free(resultCPU);
    free(tabCPU);
      
    return 0;*/
    
    float *x, *y, *d_x, *d_y;
    x = (int*)malloc(ROW_NUM*COLUMN_NUM*sizeof(int));
    y = (int*)malloc(ROW_NUM*sizeof(int));

    hipMalloc(&d_x, ROW_NUM*COLUMN_NUM*sizeof(int));
    hipMalloc(&d_y, ROW_NUM*sizeof(int));

    srand(0);
    
    for(int column=0;column<COLUMN_NUM-1;++column)
    {
        for(int row=0;row<ROW_NUM;++row)
        {
            x[ROW_NUM*column+row] = rand()%1000000;
        }
    }

    hipMemcpy(d_x, x, ROW_NUM*COLUMN_NUM*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, ROW_NUM*sizeof(int), hipMemcpyHostToDevice);

    request<<<(ROW_NUM+255)/256, 256>>>(d_x, d_y);

    hipMemcpy(y, d_y, ROW_NUM*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}